#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <time.h>

#include <hip/hip_runtime.h>


#define ASCIIMIN 32
#define ASCIIMAX 126

#define CHARS_PER_THREADS 256
#define THREADS_PER_BLOCK 256

void __global__ kernel(int nLines, char* dev_chars, int nChars, int* dev_counts, int nCounts) {
	const unsigned int tidb = threadIdx.x;
	const unsigned int ti = blockIdx.x*blockDim.x + tidb;

	__shared__ int shared_counts[ASCIIMAX - ASCIIMIN + 1];

	if (tidb == 0) {
		for (int i = 0; i < ASCIIMAX - ASCIIMIN - 1; ++i) {
			shared_counts[i] = 0;
		}
	}

	__syncthreads();

	if (ti < nLines) {
		for (int i = 0; i < CHARS_PER_THREADS; ++i) {
			int ascii = (int)dev_chars[CHARS_PER_THREADS * ti + i];
			atomicAdd(&shared_counts[ascii - ASCIIMIN], 1);
		}
	}

	__syncthreads();

	if (tidb == 0) {
		for (int i = 0; i < nCounts; ++i) {
			atomicAdd(&dev_counts[i], shared_counts[i]);
		}
	}
}

int isValid(char* c){
	int asciicode = (int)*c;
	int valid = (asciicode <= 126);
	valid = valid && (asciicode >= 32);
	return valid;
}

int main(int argc, char** argv){

	clock_t t1, t2;
	printf("Initialisation...\n");
	t1 = clock();

	//Declarations
	FILE* inputFile =  NULL;
	FILE* outputFile =  NULL;
	char* inputFileName = NULL;
	char* outputFileName = NULL;

	int nChars = 0;
	char* chars;
	int nCounts = ASCIIMAX - ASCIIMIN + 1;
	int* counts;

	char* dev_chars;
	int* dev_counts;

	int opt;

	//Get comand line options
	while ((opt = getopt (argc, argv, "i:o:")) != -1) {
		switch(opt) {
			case 'i':
				inputFileName = optarg;
				break;
			case 'o':
				outputFileName = optarg;
				break;
		}
	}

	//Count number of chars in inputFile
	inputFile = fopen(inputFileName,"r");
	if (!inputFile) return 1;
	nChars = 0;
	for (char c = getc(inputFile); c != EOF; c = getc(inputFile)){
		if(isValid(&c)) ++nChars;
	}
	fclose(inputFile);

	//Allocate memory
	counts = (int*) malloc(nCounts * sizeof(int));
	chars = (char*) malloc(nChars * sizeof(char));
	if(chars == NULL) {
		printf("Input file too large!\n");
		return 1;
	}
	printf("%d chars processed\n", nChars);
	hipMalloc( (void**)&dev_chars, nChars * sizeof(char));
	hipMalloc( (void**)&dev_counts, nCounts * sizeof(int));

	//Filling chars array
	inputFile = fopen(inputFileName,"r");
	if (!inputFile) return 1;
	int i = 0;
	for (char c = getc(inputFile); c != EOF; c = getc(inputFile)){
		if(isValid(&c)){
			if (c>=65 && c<=90) chars[i] = (char)(c + 32);
			else chars[i] = c;
			++i;
		}
	}
	fclose(inputFile);

	t1 = clock() - t1;
	printf("Process...\n");
	t2 = clock();

	//Initialize counter array
	for (int i = 0; i < nCounts; ++i){
		counts[i] = 0;
	}

	hipMemcpy(dev_chars, chars, nChars * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(dev_counts, counts, nCounts * sizeof(int), hipMemcpyHostToDevice);

	//Count chars
	for (int i = 0; i < nChars; ++i){
		int ascii = (int)chars[i];
		++counts[ascii - ASCIIMIN];
	}

	int nLines = (nChars + CHARS_PER_THREADS - 1) / CHARS_PER_THREADS;

	kernel<<<(nLines + THREADS_PER_BLOCK - 1 ) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(nLines, dev_chars, nChars, dev_counts, nCounts);

	hipMemcpy(counts, dev_counts, nCounts * sizeof(int), hipMemcpyDeviceToHost);

	t2 = clock() - t2;

	//Write in outputFile
	outputFile = fopen(outputFileName, "w+");
	if (!outputFile) return 1;
	for (int i = 0; i < 127 - ASCIIMIN; ++i){
		if (i + ASCIIMIN < 65 || i + ASCIIMIN > 90) {
			fprintf(outputFile, "%c:%d\n", (char)(i + ASCIIMIN), (int)counts[i]);
		}
	}
	fclose(outputFile);

	//Return memory
	hipFree(dev_chars);
	hipFree(dev_counts);
	free(chars);
	free(counts);

	printf("Timings:\nInitialisation: %f\nProcess:        %f\n", (float)1000 * t1/CLOCKS_PER_SEC, (float)1000 * t2/CLOCKS_PER_SEC);


	return 0;

}

